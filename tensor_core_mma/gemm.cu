// nvcc gemm.cu -o gemm -arch=compute_86 -code=sm_86 --ptxas-options=-v

#include <stdint.h>
#include <algorithm>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
// #include "cublas_v2.h"

using namespace std;

#define LDMATRIX_X1(R, addr) \
    asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))

#define LDMATRIX_X2(R0, R1, addr) \
    asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))

#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                             \
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" \
                 : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                             \
                 : "r"(addr))

#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1)                                                    \
    asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" \
                 : "=r"(RD0), "=r"(RD1)                                                                                \
                 : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

#if ((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || (__CUDACC_VER_MAJOR__ > 11)
#define CP_ASYNC_CA(dst, src, Bytes) \
    asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))
#else
#define CP_ASYNC_CA(dst, src, Bytes) \
    asm volatile("cp.async.ca.shared.global [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))

#define CP_ASYNC_CG(dst, src, Bytes) \
    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(Bytes))
#endif

#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define CP_ASYNC_WAIT_GROUP(N) asm volatile("cp.async.wait_group %0;\n" ::"n"(N))

#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16
#define WARP_SIZE 32

__global__ void mmaNaiveKernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, size_t M,
                               size_t N, size_t K) {
    const size_t K_tiles = div_ceil(K, MMA_K);

    const size_t warp_row = blockIdx.y * MMA_M;
    const size_t warp_col = blockIdx.x * MMA_N;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    __shared__ half A_smem[MMA_M][MMA_K];
    __shared__ half B_smem[MMA_N][MMA_K];
    __shared__ half C_smem[MMA_M][MMA_N];

    const size_t lane_id = threadIdx.x % WARP_SIZE;

    uint32_t RC[2] = {0, 0};

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        *((int4 *)(&A_smem[lane_id / 2][0]) + lane_id % 2) =
            *((int4 *)(&A[(warp_row + lane_id / 2) * K + i * MMA_K]) + lane_id % 2);

        if (lane_id < MMA_N * 2) {
            *((int4 *)(&B_smem[lane_id / 2][0]) + lane_id % 2) =
                *((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) + lane_id % 2);
        }

        __syncthreads();

        uint32_t RA[4];
        uint32_t RB[2];

        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_smem_lane_addr);

        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
        LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

        HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

        __syncthreads();
    }

    *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
    *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

    __syncthreads();

    if (lane_id < MMA_M) {
        *((int4 *)(&C[(warp_row + lane_id) * N + warp_col])) = *((int4 *)(&C_smem[lane_id][0]));
    }
}

void run_mmaNaive(half* a, half* b, half* c, size_t M, size_t N, size_t K) {
    half *a_d, *b_d, *c_d;
    hipMalloc((void**)&a_d, sizeof(half) * M * K);
    hipMalloc((void**)&b_d, sizeof(half) * K * N);
    hipMalloc((void**)&c_d, sizeof(half) * M * N);
    hipMemcpy(a_d, a, sizeof(half) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(half) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, sizeof(half) * M * N, hipMemcpyHostToDevice);

    dim3 blockDim(WARP_SIZE);
    dim3 gridDim(div_ceil(N, MMA_N), div_ceil(M, MMA_M));
    mmaNaiveKernel<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K);

    hipMemcpy(c, c_d, sizeof(half) * M * N, hipMemcpyDeviceToHost);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}

int main() {
    size_t M = 128;
    size_t N = 128;
    size_t K = 128;

    half *a, *b, *c;
    a = (half*)malloc(M * N * sizeof(half));
    b = (half*)malloc(M * N * sizeof(half));
    c = (half*)malloc(M * N * sizeof(half));
    for (int i = 0; i < M * N; ++i) {
        a[i] = 0.1;
        b[i] = 0.1;
        c[i] = 1.0;
    }

    run_mmaNaive(a, b, c, M, N, K);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << (float)c[i * N + j] << ' ';
        }
        cout << endl;
    }

    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;
}


